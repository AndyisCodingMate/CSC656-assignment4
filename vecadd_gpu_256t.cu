#include <iostream>
#include <chrono>
#include <unistd.h>
#include <hip/hip_runtime.h>


#define BLOCKSPerGRID 1
#define THREADSPerBLOCK 256

__global__ void add (int n, float *x, float *y) {
    int index = threadIdx.x;
    int stride = blockDim.x;

    for (int i = index; i < n; i+=stride) {
        y[i] = x[i] + y[i];
    }
}
int main(void){
   int N =  1<<29; 
   float *x, *y;
   hipHostMalloc((void**)&x, N * sizeof(float), hipHostMallocDefault);
   hipHostMalloc((void**)&y, N * sizeof(float), hipHostMallocDefault);
    // initialize x and y on the CPU
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int size = N * sizeof(float);
    float * d_x, * d_y;
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    // Run on 512M elements on the GPU
    add<<<BLOCKSPerGRID,THREADSPerBLOCK>>>(N, d_x, d_y);
    hipDeviceSynchronize();
    
    // Copy result back to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_x); hipFree(d_y);
    delete [] x; delete [] y;
    return 0;
}