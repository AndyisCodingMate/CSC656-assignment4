#include <iostream>
#include <chrono>
#include <unistd.h>
#include <hip/hip_runtime.h>


__global__ void add (int n, float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}
int main(int ac, char *av[]){
   int N =  1<<29; 
   float *x, *y;
   
   int size = N * sizeof(float);
   hipMallocManaged((void**)&x, size);
   hipMallocManaged((void**)&y, size);

    // initialize x and y on the CPU
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // Run on 512M elements on the GPU
    add<<<1,1>>>(N, x, y);
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x); hipFree(y);
    return 0;
}