#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <unistd.h>
#include <hip/hip_runtime.h>


#define BLOCKSPerGRID 1
#define THREADSPerBLOCK 1

__global__ void add (int n, float *x, float *y) {
    for (int i = 0; i < n; i++) {
        y[i] = x[i] + y[i];
    }
}
int main(void){
   int N =  1<<29; 
   hipHostMalloc((void**)&x, N * sizeof(float));
   hipHostMalloc((void**)&y, N * sizeof(float));
    // initialize x and y on the CPU
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int size = N * sizeof(float);
    float * d_x, * d_y;
    hipMalloc((void**)&d_x, size);
    hipMalloc((void**)&d_y, size);
    hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);
    
    // Run on 512M elements on the GPU
    add<<<BLOCKSPerGRID,THREADSPerBLOCK>>>(N, d_x, d_y);
    hipDeviceSynchronize();
    
    // Copy result back to host
    hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_x); hipFree(d_y);
    delete [] x; delete [] y;
    return 0;
}