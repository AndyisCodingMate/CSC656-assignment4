#include "hip/hip_runtime.h"
#include <iostream>
#include <chrono>
#include <unistd.h>
#include <hip/hip_runtime.h>

// Run kernel on GPU
#define THREADSPerBLOCK 256
int numBlocks = (N + THREADSPerBLOCK - 1) / THREADSPerBLOCK;

__global__ void add (int n, float *x, float *y) {
    int index = blockDim.x * blockIdx.x + threadIdx.x;
    
    if (index < n) {
        y[i] = x[i] + y[i];
    }
}
int main(void){
   int N =  1<<29; 
   float *x, *y;
   
   int size = N * sizeof(float);
   hipMallocManaged((void**)&x, size);
   hipMallocManaged((void**)&y, size);

    // initialize x and y on the CPU
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int numBlocks = (N + THREADSPerBLOCK - 1) / THREADSPerBLOCK;
    printf("numBlocks: %d\n", numBlocks);
    
    // Run on 512M elements on the GPU
    add<<<numBlocks,THREADSPerBLOCK>>>(N, x, y);
    hipDeviceSynchronize();

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // Free memory
    hipFree(x); hipFree(y);
    return 0;
}

